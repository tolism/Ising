//! Ising model evolution
/*!

  \param G      Spins on the square lattice             [n-by-n]
  \param w      Weight matrix                           [5-by-5]
  \param k      Number of iterations                    [scalar]
  \param n      Number of lattice points per dim        [scalar]

  NOTE: Both matrices G and w are stored in row-major format.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

#define GRID_SIZE_X 517
#define GRID_SIZE_Y 517

#define WSIZE 5

#define old(i,j,n) *(old+i*n+j)
#define current(i,j,n) *(current+i*n+j)
#define w(i,j) *(w+i*5+j)
#define d_w(i,j) *(d_w+i*5+j)
#define G(i,j,n) *(G+i*n+j)
#define d_current(i,j,n) *(d_current+i*n+j)
#define d_old(i,j,n) *(d_old+i*n+j)

void swapElement(int  ** one, int  ** two) {
  int  * temp = * one;
  * one = * two;
  * two = temp;
}

//grafika grid/block
//pou pernaw ti

__global__
   void kernel2D(int *d_current, int *d_old, double *d_w, int n , int * d_flag)
{
    // Compute column and row indices.
     int r = blockIdx.x * blockDim.x + threadIdx.x;
     int c = blockIdx.y * blockDim.y + threadIdx.y;
    //const int i = r * n + c; // 1D flat index

    double influence = 0;
    // NA VALW SIGOURA ENAN ELEGXO EDW PERA AN EIMASTE SE BOUNDS
    //  // Check if within bounds.
    if ((c >= n) || (r >= n))
        return;
    // COLUMN H ROW MAJOR AUTES OI PIPES
         for(int i = r; i<n; i+=blockDim.x*gridDim.x){
            for(int j = c; j<n; j+=blockDim.y*gridDim.y){

                  for(int ii=0; ii<5; ii++){
                    for(int jj=0; jj<5; jj++){
                      influence +=  d_w(ii,jj) * d_old((i-2+n+ii)%n, (j-2+n+jj)%n, n);
                    }
                  }
                  // magnetic moment gets the value of the SIGN of the weighted influence of its neighbors
                  if(fabs(influence) < 10e-7){
                    d_current(i,j,n) = d_old(i,j,n); // remains the same in the case that the weighted influence is zero
                  }
                  else if(influence > 10e-7){
                    d_current(i,j,n) = 1;
                    *d_flag = 0;
                  }
                  else if(influence < 0){
                    d_current(i,j,n) = -1;
                    *d_flag=0;
                  }
                  influence = 0;
                }
              }

}

void ising( int *G, double *w, int k, int n){

  dim3 block(BLOCK_SIZE_X,BLOCK_SIZE_Y);
  dim3 grid((GRID_SIZE_X+block.x-1)/block.x,(GRID_SIZE_Y+block.y - 1)/block.y);

      int * old = (int*) malloc(n*n*(size_t)sizeof(int)); // old spin lattice
      int * current = (int*) malloc(n*n*(size_t)sizeof(int)); // current spin lattice

      if(old==NULL || current == NULL){
        printf("Problem at memory allocation at host \n");
        exit(0);
      }

  int * d_old;
  int * d_current;
  double * d_w;// na valw void ** skatakia
  int *d_flag ;
  int flag ;

  if( hipMalloc((void **)&d_old ,n*n*(size_t)sizeof(int)) != hipSuccess  || hipMalloc((void **)&d_current,n*n*(size_t)sizeof(int))   != hipSuccess   || hipMalloc((void **)&d_w, WSIZE*WSIZE*(size_t)sizeof(double))   != hipSuccess || hipMalloc(&d_flag,(size_t)sizeof(int)) !=hipSuccess){
    printf("Problem at memory allocation");
    exit(0);
  }

  hipMemcpy(d_w, w, WSIZE*WSIZE*sizeof(double), hipMemcpyHostToDevice );
  hipMemcpy(d_old, G, n*n*sizeof(int), hipMemcpyHostToDevice );


  // run for k steps
  for(int l=0; l<k; l++){
    flag = 0;
    kernel2D<<<grid,block>>>(d_current, d_old, d_w, n  , d_flag );
  //  kernel2D<<<dimGrid,dimBlock>>>(d_current, d_old, d_w, n );
    hipDeviceSynchronize();

  //  cudaMemcpy(old, d_old, n*n*sizeof(int), cudaMemcpyDeviceToHost );
    hipMemcpy(current, d_current, n*n*sizeof(int), hipMemcpyDeviceToHost );
    // save result in G
    memcpy(G , current , n*n*sizeof(int));

    // swap the pointers for the next iteration
    swapElement(&d_old,&d_current);

    hipMemcpy(&flag , d_flag , (size_t)sizeof(int), hipMemcpyDeviceToHost);
    // terminate if no changes are made
    if(flag){
      printf("No changes \n");
      exit(0);
    }

  }

  free(old);
  free(current);
  hipFree(d_old);
  hipFree(d_current);
  hipFree(d_w);
}


int main(int argc, const char* argv[]){


  int n = 517;
  double weights[] = {0.004, 0.016, 0.026, 0.016, 0.004, 0.016, 0.071, 0.117, 0.071, 0.016, 0.026, 0.117, 0.0, 0.117, 0.026, 0.016, 0.071, 0.117, 0.071, 0.016, 0.004, 0.016, 0.026, 0.016, 0.004};
  int G1[n*n]; // G that changes k times
  int G2[n*n]; // G that changes k times
  int G3[n*n]; // G that changes k times
  int Gk1[n*n];
  int Gk2[n*n];
  int Gk3[n*n];
  FILE *ptr;

  // read initial G
  ptr = fopen("conf-init.bin","rb");
  fread(G1,sizeof(G1),1,ptr);
  fclose(ptr);
  // read initial G
  ptr = fopen("conf-init.bin","rb");
  fread(G2,sizeof(G2),1,ptr);
  fclose(ptr);
  // read initial G
  ptr = fopen("conf-init.bin","rb");
  fread(G3,sizeof(G3),1,ptr);
  fclose(ptr);
  // read k-th Gk
  ptr = fopen("conf-1.bin","rb"); // allazo onoma arxeiou gia allagi k
  fread(Gk1,sizeof(Gk1),1,ptr);
  fclose(ptr);

  ptr = fopen("conf-4.bin","rb"); // allazo onoma arxeiou gia allagi k
  fread(Gk2,sizeof(Gk2),1,ptr);
  fclose(ptr);

  ptr = fopen("conf-11.bin","rb"); // allazo onoma arxeiou gia allagi k
  fread(Gk3,sizeof(Gk3),1,ptr);
  fclose(ptr);

  // execution
  ising(G1, weights, 1, n);

  // check correctness
  int c = 0;
  for(int i=0; i<n; i++){
    for(int j=0; j<n; j++){
      if( *(G1+i*n+j) != *(Gk1+i*n+j) ){
        c++;
      }
    }
  }

  if(c!=0){
    printf("k=1 Wrong\n");
  }
  else{
    printf("k=1 Correct\n");
  }

  // execution
  ising(G2, weights, 4, n);

  // check correctness
   c = 0;
  for(int i=0; i<n; i++){
    for(int j=0; j<n; j++){
      if( *(G2+i*n+j) != *(Gk2+i*n+j) ){
        c++;
      }
    }
  }
  if(c!=0){
    printf("k=4 Wrong\n");
  }
  else{
    printf("k=4 Correct\n");
  }

  clock_t start = clock();

  // // execution
  // ising(G, weights, 100, 1000);
  // execution
  ising(G3, weights, 11 , n);


  clock_t end = clock();
  double exec_time = (end - start)/(double)CLOCKS_PER_SEC;

  printf("%lf\n", exec_time);



  // check correctness
   c = 0;
  for(int i=0; i<n; i++){
    for(int j=0; j<n; j++){
      if( *(G3+i*n+j) != *(Gk3+i*n+j) ){
        c++;
      }
    }
  }
  if(c!=0){
    printf("k=11 Wrong\n");
  }
  else{
    printf("k=11 Correct\n");
  }


  }
