//! Ising model evolution
/*!

  \param G      Spins on the square lattice             [n-by-n]
  \param w      Weight matrix                           [5-by-5]
  \param k      Number of iterations                    [scalar]
  \param n      Number of lattice points per dim        [scalar]

  NOTE: Both matrices G and w are stored in row-major format.
*/




#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 8

#define N 517

#define WSIZE 5
#define RADIUS 2

#define old(i,j,n) *(old+(i)*n+j)
#define current(i,j,n) *(current+(i)*n+j)
#define w(i,j) *(w+(i)*5+j)
#define d_w(i,j) *(d_w+(i)*5+j)
#define s_w(i,j) *(s_w+(i)*5+j)
#define G(i,j,n) *(G+(i)*n+j)
#define d_current(i,j,n) *(d_current+(i)*n+j)
#define d_old(i,j,n) *(d_old+(i)*n+j)
#define s_old(i,j) *(s_old + (i)*(BLOCK_SIZE_X+2*RADIUS) + j)




void swapElement(int  ** one, int  ** two) {
  int  * temp = * one;
  * one = * two;
  * two = temp;
}

__global__
   void kernel2D(int *d_current, int *d_old, double *d_w, int n){

    double influence = 0;

   int r = blockIdx.x * blockDim.x + threadIdx.x;
   int c = blockIdx.y * blockDim.y + threadIdx.y;

   int lxindex = threadIdx.x + RADIUS;
   int lyindex = threadIdx.y + RADIUS;

   int strideX = blockDim.x*gridDim.x ;
   int strideY = blockDim.y*gridDim.y ;

   __shared__ double s_w[WSIZE*WSIZE];
   __shared__ int s_old[(BLOCK_SIZE_X + 2*RADIUS)*(BLOCK_SIZE_Y + 2*RADIUS)];

   if(blockDim.x > WSIZE && blockDim.y > WSIZE ){
     //Shared weights incase we have sufficient amount of threads ( threads at X > WSIZE AND  threads at Y > WSIZE)
     if(threadIdx.x < WSIZE && threadIdx.y < WSIZE){
       s_w(threadIdx.x, threadIdx.y) = d_w(threadIdx.x, threadIdx.y);
     }
   }
   //threadIdx.x == 0  , threadIdx.y == 0 will do the job
   else{
   if(threadIdx.x == 0 && threadIdx.y == 0){
     for(int i = 0 ; i<WSIZE ; i++){
       for(int j =0; j<WSIZE ; j++){
         s_w(i,j)=d_w(i,j);
       }
     }
   }
  }
  __syncthreads();

   // // Read input elements into shared memory
   for(int i=r; i<n+RADIUS; i+= strideX){
     for(int j=c; j<n+RADIUS ; j+= strideY ){

       // window for old to shared
       s_old(lxindex,lyindex) = d_old((i+n)%n,(j+n)%n,n);
       __syncthreads();

       //stencil kata x
       if( threadIdx.y < RADIUS){
         s_old( lxindex , lyindex - RADIUS ) = d_old((i + n)%n , (j-RADIUS+n)%n , n);
         s_old(lxindex  , lyindex + BLOCK_SIZE_Y ) = d_old( (i + n )%n , (j+BLOCK_SIZE_X + n)%n , n);
       }

       //stencil kata y
       if( threadIdx.x < RADIUS){
         s_old(lxindex  - RADIUS,lyindex ) = d_old( (i-RADIUS+n)%n , (j+n)%n   , n );
         s_old( lxindex + BLOCK_SIZE_Y,lyindex ) = d_old(( i +BLOCK_SIZE_X +n)%n , (j +  n)%n     , n );
       }

       //tetragwnakia stis gwnies
       if(threadIdx.x < RADIUS && threadIdx.y < RADIUS ){
         //panw aristera
         s_old(lxindex - RADIUS , lyindex - RADIUS ) = d_old ( ( i - RADIUS + n)%n , (j-RADIUS + n )%n  , n );
         //katw deksia
         s_old(lxindex +  BLOCK_SIZE_X , lyindex + BLOCK_SIZE_Y  ) = d_old( (i  + BLOCK_SIZE_X + n  )%n , (j + BLOCK_SIZE_Y +n  )%n , n);
         //katw aristera
         s_old(lxindex - RADIUS, lyindex +  BLOCK_SIZE_Y  )= d_old ((i - RADIUS + n)%n  , (j  + BLOCK_SIZE_X )%n, n);
         //panw deksia
         s_old(lxindex + BLOCK_SIZE_X,lyindex - RADIUS  ) = d_old ((i + BLOCK_SIZE_X+n  )%n , (j -RADIUS + n )%n   , n );
       }
       __syncthreads();

       if((i<n)&&(j<n)){
          influence = 0;
           for(int ii=0; ii<WSIZE ; ii++){
             for(int jj=0 ; jj<WSIZE ; jj++){
               if((ii==2) && (jj==2))
                  continue;
            //    influence +=  d_w(ii,jj) * d_old((i-2+n+ii)%n, (j-2+n+jj)%n, n);
              influence += s_w(ii,jj) * s_old( threadIdx.x + ii,threadIdx.y +  jj );
              }
             }

           // // magnetic moment gets the value of the SIGN of the weighted influence of its neighbors
           if(fabs(influence) < 10e-7){
             d_current(i,j,n) =  s_old(lxindex, lyindex  ); // remains the same in the case that the weighted influence is zero
           }
           else if(influence > 10e-7){
             d_current(i,j,n) = 1;
           }
           else if(influence < 0){
             d_current(i,j,n) = -1;
           }
       }
      __syncthreads();
     }
   }

}


void ising( int *G, double *w, int k, int n){

    dim3 block(BLOCK_SIZE_X,BLOCK_SIZE_Y);
    dim3 grid((N+block.x-1)/block.x,(N+block.y - 1)/block.y);


  int * old = (int*) malloc(n*n*sizeof(int)); // old spin lattice
  int * current = (int*) malloc(n*n*sizeof(int)); // current spin lattice

  int * d_old;
  int * d_current;
  double * d_w;// na valw void ** skatakia
  hipMalloc(&d_old , n*n*sizeof(int));
  hipMalloc(&d_current , n*n*sizeof(int));
  hipMalloc(&d_w, WSIZE*WSIZE*sizeof(double));


    if( hipMalloc((void **)&d_old ,n*n*sizeof(int)) != hipSuccess  || hipMalloc((void **)&d_current,n*n*sizeof(int))   != hipSuccess   || hipMalloc((void **)&d_w, WSIZE*WSIZE*sizeof(double))   != hipSuccess){
      printf("Problem at memory allocation");
      exit(0);
    }

  hipMemcpy(d_w, w, WSIZE*WSIZE*sizeof(double), hipMemcpyHostToDevice );

  // Elearning tester checks the values of the G so BLOCK_SIZE_Y swaping
  // The "head" pointer it can not pass the validation
  // So we manual copy
  // swapElement(&old,&G);
  // initial values saved to old
  // for(int i=0; i<n; i++){
  //   for(int j=0; j<n; j++){
  //       old(i,j,n) = G(i,j,n);
  //   }
  // }

  hipMemcpy(d_old, G, n*n*sizeof(int), hipMemcpyHostToDevice );


  // run for k steps
  for(int l=0; l<k; l++){

    kernel2D<<<grid,block>>>(d_current, d_old, d_w, n );
  //  kernel2D<<<dimGrid,dimBlock>>>(d_current, d_old, d_w, n );
    hipDeviceSynchronize();

    //cudaMemcpy(old, d_old, n*n*sizeof(int), cudaMemcpyDeviceToHost );
    hipMemcpy(current, d_current, n*n*sizeof(int), hipMemcpyDeviceToHost );
    // save result in G
    memcpy(G , current , n*n*sizeof(int));

    // swap the pointers for the next iteration
    swapElement(&d_old,&d_current);
    // tmp = old;
    // old = current;
    // current= tmp;

    // terminate if no changes are made
    int areEqual = 0;
    for(int i=0; i<n; i++){
      for(int j=0; j<n; j++){
        if(old(i,j,n) == G(i,j,n)){
          areEqual++;
        }
      }
    }
    // termination branch
    if(areEqual == n*n){
      printf("terminated: spin values stay same (step %d)\n" , l);
      exit(0);
    }

  //  cudaMemcpy(d_old, old, n*n*sizeof(int), cudaMemcpyHostToDevice );
  //  cudaMemcpy(d_current, current, n*n*sizeof(int), cudaMemcpyHostToDevice );
  }

  free(old);
  free(current);
  hipFree(d_old);
  hipFree(d_current);
  hipFree(d_w);
}




int main(int argc, const char* argv[]){


  int n = 517;
  double weights[] = {0.004, 0.016, 0.026, 0.016, 0.004, 0.016, 0.071, 0.117, 0.071, 0.016, 0.026, 0.117, 0.0, 0.117, 0.026, 0.016, 0.071, 0.117, 0.071, 0.016, 0.004, 0.016, 0.026, 0.016, 0.004};
  int G1[n*n]; // G that changes k times
  int G2[n*n]; // G that changes k times
  int G3[n*n]; // G that changes k times
  int Gk1[n*n];
  int Gk2[n*n];
  int Gk3[n*n];
  FILE *ptr;

  // read initial G
  ptr = fopen("conf-init.bin","rb");
  fread(G1,sizeof(G1),1,ptr);
  fclose(ptr);
  // read initial G
  ptr = fopen("conf-init.bin","rb");
  fread(G2,sizeof(G2),1,ptr);
  fclose(ptr);
  // read initial G
  ptr = fopen("conf-init.bin","rb");
  fread(G3,sizeof(G3),1,ptr);
  fclose(ptr);
  // read k-th Gk
  ptr = fopen("conf-1.bin","rb"); // allazo onoma arxeiou gia allagi k
  fread(Gk1,sizeof(Gk1),1,ptr);
  fclose(ptr);

  ptr = fopen("conf-4.bin","rb"); // allazo onoma arxeiou gia allagi k
  fread(Gk2,sizeof(Gk2),1,ptr);
  fclose(ptr);

  ptr = fopen("conf-11.bin","rb"); // allazo onoma arxeiou gia allagi k
  fread(Gk3,sizeof(Gk3),1,ptr);
  fclose(ptr);

  // execution
  ising(G1, weights, 1, n);

  // check correctness
  int c = 0;
  for(int i=0; i<n; i++){
    for(int j=0; j<n; j++){
      if( *(G1+i*n+j) != *(Gk1+i*n+j) ){
        printf("(%d,%d), real:%d, ours:%d\n", i,j, *(Gk1+i*517+j) , *(G1+i*517+j));
        c++;
      }
    }
  }

  if(c!=0){
    printf("k=1 Wrong\n");
  }
  else{
    printf("k=1 Correct\n");
  }

  // execution
  ising(G2, weights, 4, n);

  // check correctness
   c = 0;
  for(int i=0; i<n; i++){
    for(int j=0; j<n; j++){
      if( *(G2+i*n+j) != *(Gk2+i*n+j) ){
        c++;
      }
    }
  }
  if(c!=0){
    printf("k=4 Wrong\n");
  }
  else{
    printf("k=4 Correct\n");
  }


  // execution
  ising(G3, weights, 11 , n);

  // check correctness
   c = 0;
  for(int i=0; i<n; i++){
    for(int j=0; j<n; j++){
      if( *(G3+i*n+j) != *(Gk3+i*n+j) ){
        c++;
      }
    }
  }
  if(c!=0){
    printf("k=11 Wrong\n");
  }
  else{
    printf("k=11 Correct\n");
  }


  }
