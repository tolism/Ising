/*
* FILE: isingV1.cu
* THMMY, 7th semester, Parallel and Distributed Systems: 3rd assignment
* Parallel Implementation  of the Ising Model
* Authors:
*   Moustaklis Apostolos, 9127, amoustakl@ece.auth.gr
*   Papadakis Charis , 9128, papadakic@ece.auth.gr
* Compile command with :
*   make all
* Run command example:
*   ./src/isingV1
* It will calculate the evolution of the ising Model
* for a given number n  of points and k steps
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>


// Defines for the block and grid calculation
#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16
#define N_X 517
#define N_Y 517

// The size of the weights
#define WSIZE 5

//Helper Defines to access easier the arrays
#define old(i,j,n) *(old+i*n+j)
#define current(i,j,n) *(current+i*n+j)
#define w(i,j) *(w+i*5+j)
#define d_w(i,j) *(d_w+i*5+j)
#define G(i,j,n) *(G+i*n+j)
#define d_current(i,j,n) *(d_current+i*n+j)
#define d_old(i,j,n) *(d_old+i*n+j)


//Functions Declaration
void swapElement(int  ** one, int  ** two);
__global__
   void kernel2D(int *d_current, int *d_old, double *d_w, int n , int * d_flag);
void ising( int *G, double *w, int k, int n);


//! Ising model evolution
/*!

  \param G      Spins on the square lattice             [n-by-n]
  \param w      Weight matrix                           [5-by-5]
  \param k      Number of iterations                    [scalar]
  \param n      Number of lattice points per dim        [scalar]

  NOTE: Both matrices G and w are stored in row-major format.
*/

void ising( int *G, double *w, int k, int n){

  //Grid and block construction
  dim3 block(BLOCK_SIZE_X,BLOCK_SIZE_Y);
  dim3 grid((N_X+block.x-1)/block.x,(N_Y+block.y - 1)/block.y);
  //Device memory allocation
  int * old = (int*) malloc(n*n*(size_t)sizeof(int)); // old spin lattice
  int * current = (int*) malloc(n*n*(size_t)sizeof(int)); // current spin lattice
  //Leak check
  if(old==NULL || current == NULL){
      printf("Problem at memory allocation at host \n");
        exit(0);
      }

  int * d_old;
  int * d_current;
  double * d_w;
  int *d_flag ;
  int flag ;
  //Host memory allocation and leak check
  if( hipMalloc((void **)&d_old ,n*n*(size_t)sizeof(int)) != hipSuccess  || hipMalloc((void **)&d_current,n*n*(size_t)sizeof(int))   != hipSuccess   || hipMalloc((void **)&d_w, WSIZE*WSIZE*(size_t)sizeof(double))   != hipSuccess || hipMalloc(&d_flag,(size_t)sizeof(int)) !=hipSuccess){
    printf("Problem at memory allocation");
    exit(0);
  }
  //Copy memory from host to device
  hipMemcpy(d_w, w, WSIZE*WSIZE*sizeof(double), hipMemcpyHostToDevice );
  hipMemcpy(d_old, G, n*n*sizeof(int), hipMemcpyHostToDevice );

  // run for k steps
  for(int l=0; l<k; l++){
    flag = 0;
    kernel2D<<<grid,block>>>(d_current, d_old, d_w, n  , d_flag );
  //  kernel2D<<<dimGrid,dimBlock>>>(d_current, d_old, d_w, n );
    hipDeviceSynchronize();

  //  cudaMemcpy(old, d_old, n*n*sizeof(int), cudaMemcpyDeviceToHost );
    hipMemcpy(current, d_current, n*n*sizeof(int), hipMemcpyDeviceToHost );
    // save result in G
    memcpy(G , current , n*n*sizeof(int));

    // swap the pointers for the next iteration
    swapElement(&d_old,&d_current);

    hipMemcpy(&flag , d_flag , (size_t)sizeof(int), hipMemcpyDeviceToHost);
    // terminate if no changes are made
    if(flag){
      printf("terminated: spin values stay same (step %d)\n" , l);
      exit(0);
    }
  }
  //Memory deallocation
  free(old);
  free(current);
  hipFree(d_old);
  hipFree(d_current);
  hipFree(d_w);
}

//Helper function to swap the pointers of the arrays
void swapElement(int  ** one, int  ** two) {
  int  * temp = * one;
  * one = * two;
  * two = temp;
}

 //The kernel function that updates the values of the ising model
__global__
void kernel2D(int *d_current, int *d_old, double *d_w, int n , int * d_flag)
{

  double influence = 0;
  // Compute column and row indices.
  const int r = blockIdx.x * blockDim.x + threadIdx.x;
  const int c = blockIdx.y * blockDim.y + threadIdx.y;


  // Check if within bounds.
  if ((c >= n) || (r >= n))
  return;

  for(int ii=0; ii<WSIZE; ii++){
    for(int jj=0; jj<WSIZE; jj++){
      influence +=  d_w(ii,jj) * d_old((r-2+n+ii)%n, (c-2+n+jj)%n, n);
    }
  }
  // magnetic moment gets the value of the SIGN of the weighted influence of its neighbors
  if(fabs(influence) < 10e-7){
    d_current(r,c,n) = d_old(r,c,n); // remains the same in the case that the weighted influence is zero
  }
  else if(influence > 10e-7){
    d_current(r,c,n) = 1;
    *d_flag=0;
  }
  else if(influence < 0){
    d_current(r,c,n) = -1;
    *d_flag=0;
  }

  influence = 0 ;

}


